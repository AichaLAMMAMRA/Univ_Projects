#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define SIZE 102400
#define MOD 102399
#define STEP 1

/* ARRAY A INITIALIZER */
void init_a(int * a)
{
    int i;
    for(i=0; i<SIZE; i++)
    {
        a[i] = 1;
    }
}

/* ARRAY B INITIALIZER */
void init_b(int * b)
{
	int i, j;

	j=0;

	for(i=0; i<SIZE-1; i++)
	{
		b[j] = i;
		j = (j+STEP)%MOD;
	}	

    b[SIZE-1] = SIZE-1;
}

/* CHECKING A VALUES */
int check_a(int * a)
{
    int i;
    int correct = 1;
	for(i=0; i<SIZE; i++)
	{
		if(a[i] != (i+1)) 
		{
         
			correct = 0;
		} 
	}	

    return correct;
}


/* CUDA FUNCTION */
__global__ void mykernel(int * a, int * b, int N)
{
    int index =  threadIdx.x;
    int i;
    for(i=index; i<N; i+=blockDim.x)
    {
        a[b[i]] = a[b[i]] + b[i];
   } 
}


int main(int argc, char * argv[])
{

	int * a = (int *)malloc(sizeof(int)*SIZE);
	int * b = (int *)malloc(sizeof(int)*SIZE);

    init_a(a);
	init_b(b);


/*  INSERT CUDA ALLOCATION AND COPY HERE */
    int * d_a, * d_b;
    hipMalloc(&d_a, sizeof(int)*SIZE);
    hipMalloc(&d_b, sizeof(int)*SIZE);	

    hipMemcpy(d_a, a, sizeof(int)*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int)*SIZE, hipMemcpyHostToDevice);

	dim3 nBlocks;
	dim3 nThperBlock;

	nBlocks.x = 1;
	
	nThperBlock.x = 1024;

    struct timeval tv_start, tv_stop;

    gettimeofday(&tv_start, NULL);

	mykernel<<< nBlocks , nThperBlock >>>(d_a, d_b, SIZE);
    hipDeviceSynchronize();
	
    gettimeofday(&tv_stop, NULL);
    
/* INSERT CUDA COPY HERE */
    hipMemcpy(a, d_a, sizeof(int)*SIZE, hipMemcpyDeviceToHost);


	int correct = check_a(a);;
	
	if(0 == correct)
	{
		printf("\n\n ******************** \n ***/!\\ ERROR /!\\ *** \n ******************** \n\n");
	}
	else
	{
		printf("\n\n ******************** \n ***** SUCCESS! ***** \n ******************** \n\n");
	}

    
    int nsec  = tv_stop.tv_sec - tv_start.tv_sec;
    int nusec = tv_stop.tv_usec - tv_start.tv_usec;

    if(nusec < 0)
    {
        nusec = nusec + 1000000;
        nsec = nsec - 1;
    }    

    printf("time = %d s %d us\n", nsec,  nusec);


	return 1;
}
