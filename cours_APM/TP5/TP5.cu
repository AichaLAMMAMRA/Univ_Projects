#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include "FreeImage.h"

#define WIDTH 1920
#define HEIGHT 1024
#define BPP 24 // Since we're outputting three 8 bit RGB values

#define BLOCK_WIDTH 32

using namespace std;

__global__ void img_copy(unsigned int* src, unsigned int* dest, unsigned width, unsigned height)
{
  int col   = threadIdx.x + blockDim.x * blockIdx.x;
  int line  = threadIdx.y + blockDim.y * blockIdx.y;

  if((col < width) && (line < height))
  {
    dest[((line * width + col) * 3) + 0] = src[((line * width + col) * 3 )+ 0];
    dest[((line * width + col) * 3) + 1] = src[((line * width + col) * 3 )+ 1];
    dest[((line * width + col) * 3) + 2] = src[((line * width + col) * 3 )+ 2];
  }
}

int main (int argc , char** argv)
{
  FreeImage_Initialise();
  const char *PathName = "lena.jpg";
  const char *PathDest = "new_lena.png";
  // load and decode a regular file
  FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);

  FIBITMAP* bitmap = FreeImage_Load(FIF_JPEG, PathName, 0);

  if(! bitmap )
    exit( 1 ); //WTF?! We can't even allocate images ? Die !

  unsigned width  = FreeImage_GetWidth(bitmap);
  unsigned height = FreeImage_GetHeight(bitmap);
  unsigned pitch  = FreeImage_GetPitch(bitmap);

  fprintf(stderr, "Processing Image of size %d x %d\n", width, height);

  unsigned int *img = (unsigned int*) malloc(sizeof(unsigned int) * 3 * width * height);

  unsigned int* cuda_src;
  hipMalloc(&cuda_src, sizeof(unsigned int) * 3 * width * height); 
  unsigned int* cuda_dest;
  hipMalloc(&cuda_dest, sizeof(unsigned int) * 3 * width * height); 

  BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      int idx = ((y * width) + x) * 3;
      img[idx + 0] = pixel[FI_RGBA_RED];
      img[idx + 1] = pixel[FI_RGBA_GREEN];
      img[idx + 2] = pixel[FI_RGBA_BLUE];
      pixel += 3;
    }
    // next line
    bits += pitch;
  }

  hipMemcpy(cuda_src, img, 3 * width * height * sizeof(unsigned int), hipMemcpyHostToDevice);

  int nbBlocksx = width / BLOCK_WIDTH;
  if(width % BLOCK_WIDTH) nbBlocksx++;

  int nbBlocksy = height / BLOCK_WIDTH;
  if(height % BLOCK_WIDTH) nbBlocksy++;

  fprintf(stderr, "(%d, %d) blocks of size (%d, %d)\n", nbBlocksx, nbBlocksy, BLOCK_WIDTH, BLOCK_WIDTH);

  dim3 gridSize(nbBlocksx, nbBlocksy);
  dim3 blockSize(BLOCK_WIDTH, BLOCK_WIDTH);

  img_copy<<<gridSize, blockSize>>>(cuda_src, cuda_dest, width, height);

  hipMemcpy(img, cuda_dest, 3 * width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);

  bits = (BYTE*)FreeImage_GetBits(bitmap);
  for ( int y =0; y<height; y++)
  {
    BYTE *pixel = (BYTE*)bits;
    for ( int x =0; x<width; x++)
    {
      RGBQUAD newcolor;

      int idx = ((y * width) + x) * 3;
      newcolor.rgbRed = img[idx + 0];
      newcolor.rgbGreen = img[idx + 1];
      newcolor.rgbBlue = img[idx + 2];

      if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
      { fprintf(stderr, "(%d, %d) Fail...\n", x, y); }

      pixel+=3;
    }
    // next line
    bits += pitch;
  }

  if( FreeImage_Save (FIF_PNG, bitmap , PathDest , 0 ))
    cout << "Image successfully saved ! " << endl ;
  FreeImage_DeInitialise(); //Cleanup !

  free(img);
  hipFree(cuda_dest);
  hipFree(cuda_src);
}
